#include "hip/hip_runtime.h"
#include "hpc_helpers/cuda_raiiwrappers.cuh"
#include "hpc_helpers/all_helpers.cuh"
#include "hpc_helpers/nvtx_markers.cuh"
#include "hpc_helpers/simple_allocation.cuh"

#include "dbdata.hpp"
#include "length_partitions.hpp"
#include "convert.cuh"
#include "blosum.hpp"
#include "types.hpp"
#include "new_kernels.cuh"


#include "kernels.cuh"

#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/scatter.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/count.h>
#include <thrust/equal.h>


#include <random>
#include <iostream>
#include <string>

template<class T>
using MyPinnedBuffer = helpers::SimpleAllocationPinnedHost<T, 0>;
template<class T>
using MyDeviceBuffer = helpers::SimpleAllocationDevice<T, 0>;

//using namespace cudasw4;

int main(int argc, char** argv){
    if(argc < 4){
        std::cout << "Usage: " << argv[0] << " querylength pseudosize pseudolength\n";
        return 0;
    }
    const int deviceId = 0;
    hipStream_t stream = 0;
    const cudasw4::SequenceLengthT queryLength = std::atoi(argv[1]);
    const int numSubjects = std::atoi(argv[2]);
    const cudasw4::SequenceLengthT pseudolength = std::atoi(argv[3]);

    const int timingLoopIters = 1;

    const int gop = -11;
    const int gex = -1;

    cudasw4::BlosumType blosumType = cudasw4::BlosumType::BLOSUM62_20;

    hipSetDevice(deviceId);

    switch(blosumType){
    case cudasw4::BlosumType::BLOSUM50_20:
        {
            const auto blosum = cudasw4::BLOSUM50_20::get1D();
            const int dim = cudasw4::BLOSUM50_20::dim;
            assert(dim == 21);
            hipMemcpyToSymbol(HIP_SYMBOL(old::cBLOSUM62_dev), &(blosum[0]), dim*dim*sizeof(char));                    
        }
        break;
    default: //cudasw4::BlosumType::BLOSUM62_20
        {
            const auto blosum = cudasw4::BLOSUM62_20::get1D();
            const int dim = cudasw4::BLOSUM62_20::dim;
            assert(dim == 21);
            hipMemcpyToSymbol(HIP_SYMBOL(old::cBLOSUM62_dev), &(blosum[0]), dim*dim*sizeof(char));
        }
        break;
    }

    setProgramWideBlosum(blosumType,{deviceId});


    const char* letters = "ARNDCQEGHILKMFPSTWYV";

    std::mt19937 gen(424242);
    std::uniform_int_distribution<> dist(0,19);
    std::string querySeq(queryLength, ' ');
    for(size_t i = 0; i < queryLength; i++){
        querySeq[i] = letters[dist(gen)];
    }


    std::vector<size_t> offsets(2);
    offsets[0] = 0;
    offsets[1] = queryLength;
    std::vector<size_t> lengths(1);
    lengths[0] = queryLength;

    const cudasw4::SequenceLengthT roundedLength = SDIV(queryLength, 128) * 128 + 128;
    MyDeviceBuffer<char> d_query(roundedLength);
    std::cout << "d_query : " << (void*)d_query.data() << ", " << roundedLength << " bytes\n";
    hipMemsetAsync(d_query.data(), 20, roundedLength, stream);
    hipMemcpyAsync(d_query.data(), querySeq.data(), queryLength, hipMemcpyDefault, stream); CUERR
    //cudasw4::NW_convert_protein_single<<<SDIV(queryLength, 128), 128, 0, stream>>>(d_query.data(), queryLength); CUERR
    thrust::transform(
        thrust::device,
        d_query.data(),
        d_query.data() + queryLength,
        d_query.data(),
        cudasw4::ConvertAA_20{}
    );

    std::vector<char> FillChar(512*16, 20);

    hipMemcpyToSymbolAsync(HIP_SYMBOL(old::constantQuery4), FillChar.data(), 512*16, 0, hipMemcpyHostToDevice, stream); CUERR
    hipMemcpyToSymbolAsync(HIP_SYMBOL(old::constantQuery4), d_query.data(), queryLength, 0, hipMemcpyDeviceToDevice, stream); CUERR

    // SINGLE PASS half 2 BENCHMARKS

    #if 1
        std::cout << "NW_local_affine_Protein_single_pass_half2\n";

        //for(int pseudodbSeqLength : {64, 128, 192, 256, 320, 384, 448, 512, 576, 640, 704, 768, 832, 896, 960, 1024}){
        //for(int pseudodbSeqLength : {512}){
        //for(int pseudodbSeqLength = 11; pseudodbSeqLength <= 64; pseudodbSeqLength++){
        {
            const int pseudodbSeqLength = pseudolength;
            std::cout << "pseudodbSeqLength: " << pseudodbSeqLength << "\n";
    
            cudasw4::PseudoDB fullDB = cudasw4::loadPseudoDB(numSubjects, pseudodbSeqLength);
            const auto& dbData = fullDB.getData();

            std::vector<MyDeviceBuffer<float>> d_scores_vec(std::max(2, timingLoopIters));
            std::vector<MyDeviceBuffer<cudasw4::ReferenceIdT>> d_overflow_positions_vec_reft(std::max(2, timingLoopIters));
            std::vector<MyDeviceBuffer<size_t>> d_overflow_positions_vec_sizet(std::max(2, timingLoopIters));
            std::vector<MyDeviceBuffer<int>> d_overflow_number_vec(std::max(2, timingLoopIters));
            for(int i = 0; i < std::max(2, timingLoopIters); i++){
                d_scores_vec[i].resize(numSubjects);
                d_overflow_positions_vec_reft[i].resize(numSubjects);
                d_overflow_positions_vec_sizet[i].resize(numSubjects);
                d_overflow_number_vec[i].resize(1);
                hipMemsetAsync(d_overflow_number_vec[i].data(), 0, sizeof(int), stream);
            }

            MyDeviceBuffer<cudasw4::ReferenceIdT> d_selectedPositions_reft(numSubjects);
            MyDeviceBuffer<size_t> d_selectedPositions_sizet(numSubjects);
            thrust::sequence(thrust::cuda::par_nosync.on(stream), d_selectedPositions_reft.begin(), d_selectedPositions_reft.end(), cudasw4::ReferenceIdT(0));
            thrust::sequence(thrust::cuda::par_nosync.on(stream), d_selectedPositions_sizet.begin(), d_selectedPositions_sizet.end(), size_t(0));

            MyDeviceBuffer<char> d_subjects(dbData.numChars());
            MyDeviceBuffer<size_t> d_subjectOffsets(numSubjects+1);
            MyDeviceBuffer<cudasw4::SequenceLengthT> d_subjectLengths_lengtht(numSubjects);
            MyDeviceBuffer<size_t> d_subjectLengths_sizet(numSubjects);

            hipMemcpyAsync(d_subjects.data(), dbData.chars(), dbData.numChars(), H2D, stream); CUERR;
            hipMemcpyAsync(d_subjectOffsets.data(), dbData.offsets(), sizeof(size_t) * (numSubjects+1), H2D, stream); CUERR;
            hipMemcpyAsync(d_subjectLengths_lengtht.data(), dbData.lengths(), sizeof(cudasw4::SequenceLengthT) * numSubjects, H2D, stream); CUERR;

            thrust::copy(
                thrust::cuda::par.on(stream),
                d_subjectLengths_lengtht.data(),
                d_subjectLengths_lengtht.data() + numSubjects,
                d_subjectLengths_sizet.data()
            );

            auto checkIfEqualResultsNew = [&](){
                const float overflowscore = 123456;
                auto overflowiter = thrust::make_constant_iterator(overflowscore);
                for(int i = 0; i < 2; i++){
                    int numOverflow = 0;
                    hipMemcpyAsync(&numOverflow, d_overflow_number_vec[i].data(), sizeof(int), D2H, stream); CUERR;
                    hipStreamSynchronize(stream); CUERR;
                    // if(i == 0){
                    //     std::cout << "Num overflows: " << numOverflow << "\n";
                    // }
                    thrust::scatter(
                        thrust::cuda::par_nosync.on(stream),
                        overflowiter,
                        overflowiter + numOverflow,
                        d_overflow_positions_vec_reft[i].data(),
                        d_scores_vec[i].data()
                    );
                }
                for(int i = 1; i < 2; i++){
                    bool equal = thrust::equal(
                        thrust::cuda::par_nosync.on(stream),
                        d_scores_vec[i].data(),
                        d_scores_vec[i].data() + numSubjects,
                        d_scores_vec[0].data()
                    );
                    if(!equal){
                        std::cout << "i = " << i << ", scores not equal\n";
                    }else{
                        std::cout << "ok\n";
                    }
                }
            };

            const double timingCups = ((double(queryLength) * pseudodbSeqLength * numSubjects)) * timingLoopIters;

            using GCUPSstats = std::tuple<double, int, int, int>;

            std::vector<GCUPSstats> gcupsVec;

            #define runSinglePassHalf2(blocksize, groupsize, numRegs){ \
                assert(blocksize % groupsize == 0); \
                if(pseudodbSeqLength <= groupsize * numRegs){ \
                    constexpr int alignmentsPerBlock = (blocksize / groupsize) * 2; \
                    helpers::GpuTimer timer1(stream, "Timer_" + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                    for(int i = 0; i < timingLoopIters; i++){ \
                        old::NW_local_affine_Protein_single_pass_half2<groupsize, numRegs><<<SDIV(numSubjects, alignmentsPerBlock), blocksize, 0, stream>>>( \
                            d_subjects.data(),  \
                            d_scores_vec[i].data(),  \
                            d_subjectOffsets.data(),  \
                            d_subjectLengths_sizet.data(),  \
                            d_selectedPositions_sizet.data(),  \
                            numSubjects,  \
                            d_overflow_positions_vec_sizet[i].data(),  \
                            d_overflow_number_vec[i].data(),  \
                            1,  \
                            queryLength,  \
                            gop,  \
                            gex \
                        ); CUERR \
                    } \
                    double gcups = timingCups / 1000. / 1000. / 1000.; \
                    gcups = gcups / (timer1.elapsed() / 1000); \
                    gcupsVec.push_back(std::make_tuple(gcups,blocksize,groupsize, numRegs )); \
                } \
            }
            #define runSinglePassHalf2_new(blocksize, groupsize, numRegs){ \
                assert(blocksize % groupsize == 0); \
                const char4* d_query4 = (const char4*)d_query.data(); \
                if(pseudodbSeqLength <= groupsize * numRegs){ \
                    helpers::GpuTimer timer1(stream, "Timer_" + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                    for(int i = 0; i < timingLoopIters; i++){ \
                        cudasw4::call_NW_local_affine_Protein_single_pass_half2_new<blocksize, groupsize, numRegs>( \
                            blosumType, \
                            d_subjects.data(),  \
                            d_scores_vec[i].data(),  \
                            d_subjectOffsets.data(),  \
                            d_subjectLengths_lengtht.data(),  \
                            d_selectedPositions_reft.data(),  \
                            numSubjects,  \
                            d_overflow_positions_vec_reft[i].data(),  \
                            d_overflow_number_vec[i].data(),  \
                            0,  \
                            d_query4, \
                            queryLength,  \
                            gop,  \
                            gex, \
                            stream \
                        ); CUERR \
                    } \
                    double gcups = timingCups / 1000. / 1000. / 1000.; \
                    gcups = gcups / (timer1.elapsed() / 1000); \
                    gcupsVec.push_back(std::make_tuple(gcups,blocksize,groupsize, numRegs )); \
                } \
            }

            #define compareSinglePassHalf2New(blocksize, groupsize, numRegs){ \
                assert(blocksize % groupsize == 0); \
                const char4* d_query4 = (const char4*)d_query.data(); \
                constexpr int alignmentsPerBlock = (blocksize / groupsize) * 2; \
                helpers::GpuTimer timer1(stream, "old " + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                old::NW_local_affine_Protein_single_pass_half2<groupsize, numRegs><<<SDIV(numSubjects, alignmentsPerBlock), blocksize, 0, stream>>>( \
                    d_subjects.data(),  \
                    d_scores_vec[0].data(),  \
                    d_subjectOffsets.data(),  \
                    d_subjectLengths_sizet.data(),  \
                    d_selectedPositions_sizet.data(),  \
                    numSubjects,  \
                    d_overflow_positions_vec_sizet[0].data(),  \
                    d_overflow_number_vec[0].data(),  \
                    0,  \
                    queryLength,  \
                    gop,  \
                    gex \
                ); CUERR \
                timer1.printGCUPS(((double(queryLength) * pseudodbSeqLength * numSubjects)));\
                helpers::GpuTimer timer2(stream, "new " + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                cudaws4::call_NW_local_affine_Protein_single_pass_half2_new<blocksize, groupsize, numRegs>( \
                    blosumType, \
                    d_subjects.data(),  \
                    d_scores_vec[1].data(),  \
                    d_subjectOffsets.data(),  \
                    d_subjectLengths_lengtht.data(),  \
                    d_selectedPositions_reft.data(),  \
                    numSubjects,  \
                    d_overflow_positions_vec_reft[1].data(),  \
                    d_overflow_number_vec[1].data(),  \
                    0,  \
                    d_query4, \
                    queryLength,  \
                    gop,  \
                    gex, \
                    stream \
                ); CUERR \
                timer2.printGCUPS(((double(queryLength) * pseudodbSeqLength * numSubjects)));\
                checkIfEqualResultsNew(); \
            }

            #define runSinglePassHalf2_numregs(blocksize, numRegs){ \
                runSinglePassHalf2(blocksize, 1, numRegs); \
                runSinglePassHalf2(blocksize, 2, numRegs); \
                runSinglePassHalf2(blocksize, 4, numRegs); \
                runSinglePassHalf2(blocksize, 8, numRegs); \
                runSinglePassHalf2(blocksize, 16, numRegs); \
                runSinglePassHalf2(blocksize, 32, numRegs); \
            }
            #define runSinglePassHalf2_numregs_new(blocksize, numRegs){ \
                runSinglePassHalf2_new(blocksize, 1, numRegs); \
                runSinglePassHalf2_new(blocksize, 2, numRegs); \
                runSinglePassHalf2_new(blocksize, 4, numRegs); \
                runSinglePassHalf2_new(blocksize, 8, numRegs); \
                runSinglePassHalf2_new(blocksize, 16, numRegs); \
                runSinglePassHalf2_new(blocksize, 32, numRegs); \
            }

        // compareSinglePassHalf2New(256, 32, 32);

            // runSinglePassHalf2_numregs(256, 32);
            // runSinglePassHalf2_numregs(256, 30);
            // runSinglePassHalf2_numregs(256, 28);
            // runSinglePassHalf2_numregs(256, 26);
            // runSinglePassHalf2_numregs(256, 24);
            // runSinglePassHalf2_numregs(256, 22);
            // runSinglePassHalf2_numregs(256, 20);
            // runSinglePassHalf2_numregs(256, 18);
            // runSinglePassHalf2_numregs(256, 16);
            // runSinglePassHalf2_numregs(256, 14);
            // runSinglePassHalf2_numregs(256, 12);
            // runSinglePassHalf2_numregs(256, 10);
            // runSinglePassHalf2_numregs(256, 8);
            // runSinglePassHalf2_numregs(256, 6);
            // runSinglePassHalf2_numregs(256, 4);
            // runSinglePassHalf2_numregs(256, 2);

            runSinglePassHalf2(256, 16, 32);

            std::sort(gcupsVec.begin(), gcupsVec.end(), [](const auto& l, const auto& r){ return std::get<0>(l) > std::get<0>(r);});

            std::cout << "old\n";
            for(int i = 0; i < std::min(3, int(gcupsVec.size())); i++){
                GCUPSstats data = gcupsVec[i];
                std::cout << std::get<0>(data) << " GCUPS, " << std::get<1>(data) << " " << std::get<2>(data) << " " << std::get<3>(data) << "\n";
            }
            gcupsVec.clear();

            // runSinglePassHalf2_numregs_new(256, 32);
            // runSinglePassHalf2_numregs_new(256, 30);
            // runSinglePassHalf2_numregs_new(256, 28);
            // runSinglePassHalf2_numregs_new(256, 26);
            // runSinglePassHalf2_numregs_new(256, 24);
            // runSinglePassHalf2_numregs_new(256, 22);
            // runSinglePassHalf2_numregs_new(256, 20);
            // runSinglePassHalf2_numregs_new(256, 18);
            // runSinglePassHalf2_numregs_new(256, 16);
            // runSinglePassHalf2_numregs_new(256, 14);
            // runSinglePassHalf2_numregs_new(256, 12);
            // runSinglePassHalf2_numregs_new(256, 10);
            // runSinglePassHalf2_numregs_new(256, 8);
            // runSinglePassHalf2_numregs_new(256, 6);
            // runSinglePassHalf2_numregs_new(256, 4);
            // runSinglePassHalf2_numregs_new(256, 2);

            runSinglePassHalf2_new(256, 16, 32);

            std::sort(gcupsVec.begin(), gcupsVec.end(), [](const auto& l, const auto& r){ return std::get<0>(l) > std::get<0>(r);});

            std::cout << "new\n";
            for(int i = 0; i < std::min(3, int(gcupsVec.size())); i++){
                GCUPSstats data = gcupsVec[i];
                std::cout << std::get<0>(data) << " GCUPS, " << std::get<1>(data) << " " << std::get<2>(data) << " " << std::get<3>(data) << "\n";
            }
            gcupsVec.clear();
        }

    #endif




    // MANY PASS HALF2 BENCHMARKS

    #if 1

        std::cout << "NW_local_affine_Protein_many_pass_half2\n";

        //for(int pseudodbSeqLength : {1500, 2000, 2048, 3333, 4096, 6666, 7000}){
        //for(int pseudodbSeqLength : {4096}){
        //for(int pseudodbSeqLength = 1024+256; pseudodbSeqLength <= 8192; pseudodbSeqLength += 256){
        {
            const int pseudodbSeqLength = pseudolength;
            std::cout << "pseudodbSeqLength: " << pseudodbSeqLength << "\n";
    
            cudasw4::PseudoDB fullDB = cudasw4::loadPseudoDB(numSubjects, pseudodbSeqLength);
            const auto& dbData = fullDB.getData();

            std::vector<MyDeviceBuffer<float>> d_scores_vec(std::max(2, timingLoopIters));
            std::vector<MyDeviceBuffer<cudasw4::ReferenceIdT>> d_overflow_positions_vec_reft(std::max(2, timingLoopIters));
            std::vector<MyDeviceBuffer<size_t>> d_overflow_positions_vec_sizet(std::max(2, timingLoopIters));
            std::vector<MyDeviceBuffer<int>> d_overflow_number_vec(std::max(2, timingLoopIters));
            for(int i = 0; i < std::max(2, timingLoopIters); i++){
                d_scores_vec[i].resize(numSubjects);
                d_overflow_positions_vec_reft[i].resize(numSubjects);
                d_overflow_positions_vec_sizet[i].resize(numSubjects);
                d_overflow_number_vec[i].resize(1);
                hipMemsetAsync(d_overflow_number_vec[i].data(), 0, sizeof(int), stream);
            }

            MyDeviceBuffer<cudasw4::ReferenceIdT> d_selectedPositions_reft(numSubjects);
            MyDeviceBuffer<size_t> d_selectedPositions_sizet(numSubjects);
            thrust::sequence(thrust::cuda::par_nosync.on(stream), d_selectedPositions_reft.begin(), d_selectedPositions_reft.end(), cudasw4::ReferenceIdT(0));
            thrust::sequence(thrust::cuda::par_nosync.on(stream), d_selectedPositions_sizet.begin(), d_selectedPositions_sizet.end(), size_t(0));

            MyDeviceBuffer<char> d_subjects(dbData.numChars());
            MyDeviceBuffer<size_t> d_subjectOffsets(numSubjects+1);
            MyDeviceBuffer<cudasw4::SequenceLengthT> d_subjectLengths_lengtht(numSubjects);
            MyDeviceBuffer<size_t> d_subjectLengths_sizet(numSubjects);

            hipMemcpyAsync(d_subjects.data(), dbData.chars(), dbData.numChars(), H2D, stream); CUERR;
            hipMemcpyAsync(d_subjectOffsets.data(), dbData.offsets(), sizeof(size_t) * (numSubjects+1), H2D, stream); CUERR;
            hipMemcpyAsync(d_subjectLengths_lengtht.data(), dbData.lengths(), sizeof(cudasw4::SequenceLengthT) * numSubjects, H2D, stream); CUERR;

            thrust::copy(
                thrust::cuda::par.on(stream),
                d_subjectLengths_lengtht.data(),
                d_subjectLengths_lengtht.data() + numSubjects,
                d_subjectLengths_sizet.data()
            );

            MyDeviceBuffer<__half2> d_tempH(size_t(queryLength) * SDIV(numSubjects, 64) * 64);
            MyDeviceBuffer<__half2> d_tempE(size_t(queryLength) * SDIV(numSubjects, 64) * 64);

            const double timingCups = ((double(queryLength) * pseudodbSeqLength * numSubjects)) * timingLoopIters;

            using GCUPSstats = std::tuple<double, int, int, int>;

            std::vector<GCUPSstats> gcupsVec;

            auto checkIfEqualResults = [&](){
                const float overflowscore = 123456;
                auto overflowiter = thrust::make_constant_iterator(overflowscore);
                for(int i = 0; i < timingLoopIters; i++){
                    int numOverflow = 0;
                    hipMemcpyAsync(&numOverflow, d_overflow_number_vec[i].data(), sizeof(int), D2H, stream); CUERR;
                    hipStreamSynchronize(stream); CUERR;
                    // if(i == 0){
                    //     std::cout << "Num overflows: " << numOverflow << "\n";
                    // }
                    thrust::scatter(
                        thrust::cuda::par_nosync.on(stream),
                        overflowiter,
                        overflowiter + numOverflow,
                        d_overflow_positions_vec_reft[i].data(),
                        d_scores_vec[i].data()
                    );
                }
                for(int i = 1; i < timingLoopIters; i++){
                    bool equal = thrust::equal(
                        thrust::cuda::par_nosync.on(stream),
                        d_scores_vec[i].data(),
                        d_scores_vec[i].data() + numSubjects,
                        d_scores_vec[0].data()
                    );
                    if(!equal){
                        std::cout << "i = " << i << ", scores not equal\n";
                    }
                }
            };

            auto checkIfEqualResultsNew = [&](){
                const float overflowscore = 123456;
                auto overflowiter = thrust::make_constant_iterator(overflowscore);
                for(int i = 0; i < 2; i++){
                    int numOverflow = 0;
                    hipMemcpyAsync(&numOverflow, d_overflow_number_vec[i].data(), sizeof(int), D2H, stream); CUERR;
                    hipStreamSynchronize(stream); CUERR;
                    // if(i == 0){
                    //     std::cout << "Num overflows: " << numOverflow << "\n";
                    // }
                    thrust::scatter(
                        thrust::cuda::par_nosync.on(stream),
                        overflowiter,
                        overflowiter + numOverflow,
                        d_overflow_positions_vec_reft[i].data(),
                        d_scores_vec[i].data()
                    );
                }
                for(int i = 1; i < 2; i++){
                    bool equal = thrust::equal(
                        thrust::cuda::par_nosync.on(stream),
                        d_scores_vec[i].data(),
                        d_scores_vec[i].data() + numSubjects,
                        d_scores_vec[0].data()
                    );
                    if(!equal){
                        std::cout << "i = " << i << ", scores not equal\n";
                    }else{
                        std::cout << "ok\n";
                    }
                }
            };

            #define runManyPassHalf2(blocksize, groupsize, numRegs){ \
                assert(blocksize % groupsize == 0); \
                constexpr int alignmentsPerBlock = (blocksize / groupsize) * 2; \
                helpers::GpuTimer timer1(stream, "Timer_" + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                for(int i = 0; i < timingLoopIters; i++){ \
                    hipMemsetAsync(d_tempH.data(), 0, d_tempH.size() * sizeof(__half2), stream); CUERR; \
                    hipMemsetAsync(d_tempE.data(), 0, d_tempE.size() * sizeof(__half2), stream); CUERR; \
                    old::NW_local_affine_Protein_many_pass_half2<groupsize, numRegs><<<SDIV(numSubjects, alignmentsPerBlock), blocksize, 0, stream>>>( \
                        d_subjects.data(),  \
                        d_scores_vec[i].data(),  \
                        d_tempH.data(), \
                        d_tempE.data(), \
                        d_subjectOffsets.data(),  \
                        d_subjectLengths_sizet.data(),  \
                        d_selectedPositions_sizet.data(),  \
                        numSubjects,  \
                        d_overflow_positions_vec_sizet[i].data(),  \
                        d_overflow_number_vec[i].data(),  \
                        0,  \
                        queryLength,  \
                        gop,  \
                        gex \
                    ); CUERR \
                } \
                timer1.stop(); \
                double gcups = timingCups / 1000. / 1000. / 1000.; \
                gcups = gcups / (timer1.elapsed() / 1000); \
                gcupsVec.push_back(std::make_tuple(gcups,blocksize,groupsize, numRegs )); \
            }

                    //hipMemsetAsync(d_tempH.data(), 0, d_tempH.size() * sizeof(__half2), stream); CUERR; 
                    //hipMemsetAsync(d_tempE.data(), 0, d_tempE.size() * sizeof(__half2), stream); CUERR; 
            #define runManyPassHalf2_new(blocksize, groupsize, numRegs){ \
                assert(blocksize % groupsize == 0); \
                const char4* d_query4 = (const char4*)d_query.data(); \
                helpers::GpuTimer timer1(stream, "Timer_" + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                for(int i = 0; i < timingLoopIters; i++){ \
                    cudasw4::call_NW_local_affine_Protein_many_pass_half2_new<blocksize, groupsize, numRegs>( \
                        blosumType, \
                        d_subjects.data(),  \
                        d_scores_vec[i].data(),  \
                        d_tempH.data(), \
                        d_tempE.data(), \
                        d_subjectOffsets.data(),  \
                        d_subjectLengths_lengtht.data(),  \
                        d_selectedPositions_reft.data(),  \
                        numSubjects,  \
                        d_overflow_positions_vec_reft[i].data(),  \
                        d_overflow_number_vec[i].data(),  \
                        0,  \
                        d_query4, \
                        queryLength,  \
                        gop,  \
                        gex, \
                        stream \
                    ); CUERR \
                } \
                timer1.stop(); \
                double gcups = timingCups / 1000. / 1000. / 1000.; \
                gcups = gcups / (timer1.elapsed() / 1000); \
                gcupsVec.push_back(std::make_tuple(gcups,blocksize,groupsize, numRegs )); \
            }

            #define compareManyPassHalf2New(blocksize, groupsize, numRegs){ \
                assert(blocksize % groupsize == 0); \
                const char4* d_query4 = (const char4*)d_query.data(); \
                constexpr int alignmentsPerBlock = (blocksize / groupsize) * 2; \
                    hipMemsetAsync(d_tempH.data(), 0, d_tempH.size() * sizeof(__half2), stream); CUERR; \
                    hipMemsetAsync(d_tempE.data(), 0, d_tempE.size() * sizeof(__half2), stream); CUERR; \
                    helpers::GpuTimer timer1(stream, "old " + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                    old::NW_local_affine_Protein_many_pass_half2<groupsize, numRegs><<<SDIV(numSubjects, alignmentsPerBlock), blocksize, 0, stream>>>( \
                        d_subjects.data(),  \
                        d_scores_vec[0].data(),  \
                        d_tempH.data(), \
                        d_tempE.data(), \
                        d_subjectOffsets.data(),  \
                        d_subjectLengths_sizet.data(),  \
                        d_selectedPositions_sizet.data(),  \
                        numSubjects,  \
                        d_overflow_positions_vec_sizet[0].data(),  \
                        d_overflow_number_vec[0].data(),  \
                        0,  \
                        queryLength,  \
                        gop,  \
                        gex \
                    ); CUERR \
                    timer1.printGCUPS(((double(queryLength) * pseudodbSeqLength * numSubjects)));\
                    hipMemsetAsync(d_tempH.data(), 0, d_tempH.size() * sizeof(__half2), stream); CUERR; \
                    hipMemsetAsync(d_tempE.data(), 0, d_tempE.size() * sizeof(__half2), stream); CUERR; \
                    helpers::GpuTimer timer2(stream, "new " + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                    cudasw4::call_NW_local_affine_Protein_many_pass_half2_new<groupsize, numRegs>( \
                        blosumType, \
                        d_subjects.data(),  \
                        d_scores_vec[1].data(),  \
                        d_tempH.data(), \
                        d_tempE.data(), \
                        d_subjectOffsets.data(),  \
                        d_subjectLengths_lengtht.data(),  \
                        d_selectedPositions_reft.data(),  \
                        numSubjects,  \
                        d_overflow_positions_vec_reft[1].data(),  \
                        d_overflow_number_vec[1].data(),  \
                        0,  \
                        d_query4, \
                        queryLength,  \
                        gop,  \
                        gex, \
                        stream \
                    ); CUERR \
                    timer2.printGCUPS(((double(queryLength) * pseudodbSeqLength * numSubjects)));\
                checkIfEqualResultsNew(); \
            }


            #define runManyPassHalf2_numregs(blocksize, numRegs){ \
                runManyPassHalf2(blocksize, 1, numRegs); \
                runManyPassHalf2(blocksize, 2, numRegs); \
                runManyPassHalf2(blocksize, 4, numRegs); \
                runManyPassHalf2(blocksize, 8, numRegs); \
                runManyPassHalf2(blocksize, 16, numRegs); \
                runManyPassHalf2(blocksize, 32, numRegs); \
            }

            //runManyPassHalf2(256, 32, 32);

            //compareManyPassHalf2New(256, 32, 32);

            // std::cout << "start 4\n"; runManyPassHalf2(256, 32, 4);
            // std::cout << "start 6\n"; runManyPassHalf2(256, 32, 6);
            // std::cout << "start 8\n"; runManyPassHalf2(256, 32, 8);
            // std::cout << "start 10\n"; runManyPassHalf2(256, 32, 10);
            // std::cout << "start 12\n"; runManyPassHalf2(256, 32, 12);
            // std::cout << "start 14\n"; runManyPassHalf2(256, 32, 14);
            // std::cout << "start 16\n"; runManyPassHalf2(256, 32, 16);
            // runManyPassHalf2(256, 32, 2);
            // runManyPassHalf2(256, 32, 4);
            // runManyPassHalf2(256, 32, 6);
            // runManyPassHalf2(256, 32, 8);
            // runManyPassHalf2(256, 32, 10);
            // runManyPassHalf2(256, 32, 12);
            runManyPassHalf2(256, 32, 14);
            runManyPassHalf2(256, 32, 16);
            runManyPassHalf2(256, 32, 18);
            // runManyPassHalf2(256, 32, 20);
            // runManyPassHalf2(256, 32, 22);
            // runManyPassHalf2(256, 32, 24);
            // runManyPassHalf2(256, 32, 26);
            // runManyPassHalf2(256, 32, 28);
            // runManyPassHalf2(256, 32, 30);
            // runManyPassHalf2(256, 32, 32);

            std::sort(gcupsVec.begin(), gcupsVec.end(), [](const auto& l, const auto& r){ return std::get<0>(l) > std::get<0>(r);});
            //for(int i = 0; i < std::min(3, int(gcupsVec.size())); i++){
            for(int i = 0; i < int(gcupsVec.size()); i++){
                GCUPSstats data = gcupsVec[i];
                std::cout << std::get<0>(data) << " GCUPS, " << std::get<1>(data) << " " << std::get<2>(data) << " " << std::get<3>(data) << "\n";
            }
            std::cout << "\n";
            gcupsVec.clear();


            // runManyPassHalf2_new(256, 32, 6);
            // runManyPassHalf2_new(256, 32, 8);
            // runManyPassHalf2_new(256, 32, 10);
            // runManyPassHalf2_new(256, 32, 12);
            runManyPassHalf2_new(256, 32, 14);
            runManyPassHalf2_new(256, 32, 16);
            runManyPassHalf2_new(256, 32, 18);
            // runManyPassHalf2_new(256, 32, 20);
            // runManyPassHalf2_new(256, 32, 22);
            // runManyPassHalf2_new(256, 32, 24);
            // runManyPassHalf2_new(256, 32, 26);
            // runManyPassHalf2_new(256, 32, 28);
            // runManyPassHalf2_new(256, 32, 30);
            // runManyPassHalf2_new(256, 32, 32);

            std::sort(gcupsVec.begin(), gcupsVec.end(), [](const auto& l, const auto& r){ return std::get<0>(l) > std::get<0>(r);});
            //for(int i = 0; i < std::min(3, int(gcupsVec.size())); i++){
            for(int i = 0; i < int(gcupsVec.size()); i++){
                GCUPSstats data = gcupsVec[i];
                std::cout << std::get<0>(data) << " GCUPS, " << std::get<1>(data) << " " << std::get<2>(data) << " " << std::get<3>(data) << "\n";
            }
            std::cout << "\n";
            gcupsVec.clear();
        }

    #endif






    // MANY PASS FLOAT BENCHMARKS

    #if 0

        std::cout << "NW_local_affine_read4_float_query_Protein\n";

        //for(int pseudodbSeqLength : {1500, 2000, 2048, 3333, 4096, 6666, 7000}){
        //for(int pseudodbSeqLength : {4096})
        {
        //for(int pseudodbSeqLength = 1024+256; pseudodbSeqLength <= 8192; pseudodbSeqLength += 256){
            const int pseudodbSeqLength = pseudolength;
            std::cout << "pseudodbSeqLength: " << pseudodbSeqLength << "\n";
    
            cudasw4::PseudoDB fullDB = cudasw4::loadPseudoDB(numSubjects, pseudodbSeqLength);
            const auto& dbData = fullDB.getData();

            std::vector<MyDeviceBuffer<float>> d_scores_vec(std::max(2, timingLoopIters));
            for(int i = 0; i < std::max(2, timingLoopIters); i++){
                d_scores_vec[i].resize(numSubjects);
            }
            MyDeviceBuffer<cudasw4::ReferenceIdT> d_selectedPositions_reft(numSubjects);
            MyDeviceBuffer<size_t> d_selectedPositions_sizet(numSubjects);
            thrust::sequence(thrust::cuda::par_nosync.on(stream), d_selectedPositions_reft.begin(), d_selectedPositions_reft.end(), cudasw4::ReferenceIdT(0));
            thrust::sequence(thrust::cuda::par_nosync.on(stream), d_selectedPositions_sizet.begin(), d_selectedPositions_sizet.end(), size_t(0));

            MyDeviceBuffer<char> d_subjects(dbData.numChars());
            MyDeviceBuffer<size_t> d_subjectOffsets(numSubjects+1);
            MyDeviceBuffer<cudasw4::SequenceLengthT> d_subjectLengths_lengtht(numSubjects);
            MyDeviceBuffer<size_t> d_subjectLengths_sizet(numSubjects);

            hipMemcpyAsync(d_subjects.data(), dbData.chars(), dbData.numChars(), H2D, stream); CUERR;
            hipMemcpyAsync(d_subjectOffsets.data(), dbData.offsets(), sizeof(size_t) * (numSubjects+1), H2D, stream); CUERR;
            hipMemcpyAsync(d_subjectLengths_lengtht.data(), dbData.lengths(), sizeof(cudasw4::SequenceLengthT) * numSubjects, H2D, stream); CUERR;

            thrust::copy(
                thrust::cuda::par.on(stream),
                d_subjectLengths_lengtht.data(),
                d_subjectLengths_lengtht.data() + numSubjects,
                d_subjectLengths_sizet.data()
            );

            MyDeviceBuffer<float2> d_tempH(size_t(queryLength) * SDIV(numSubjects, 64) * 64);
            MyDeviceBuffer<float2> d_tempE(size_t(queryLength) * SDIV(numSubjects, 64) * 64);

            const double timingCups = ((double(queryLength) * pseudodbSeqLength * numSubjects)) * timingLoopIters;

            using GCUPSstats = std::tuple<double, int, int, int>;

            std::vector<GCUPSstats> gcupsVec;

            auto checkIfEqualResults = [&](){
                for(int i = 1; i < timingLoopIters; i++){
                    bool equal = thrust::equal(
                        thrust::cuda::par_nosync.on(stream),
                        d_scores_vec[i].data(),
                        d_scores_vec[i].data() + numSubjects,
                        d_scores_vec[0].data()
                    );
                    if(!equal){
                        std::cout << "i = " << i << ", scores not equal\n";
                    }
                }
            };

            auto checkIfEqualResultsNew = [&](){
                for(int i = 1; i < 2; i++){
                    float s1 = 0;
                    hipMemcpy(&s1, d_scores_vec[i].data(), sizeof(float), D2H);
                    float s0 = 0;
                    hipMemcpy(&s0, d_scores_vec[0].data(), sizeof(float), D2H);
                    std::cout << "s0 " << s0 << " " << "s1 " << s1 << "\n";
                    bool equal = thrust::equal(
                        thrust::cuda::par_nosync.on(stream),
                        d_scores_vec[i].data(),
                        d_scores_vec[i].data() + numSubjects,
                        d_scores_vec[0].data()
                    );
                    if(!equal){
                        std::cout << "i = " << i << ", scores not equal\n";
                    }else{
                        std::cout << "ok\n";
                    }
                }
            };

            #define runManyPassFloat(blocksize, groupsize, numRegs){ \
                assert(blocksize % groupsize == 0); \
                constexpr int alignmentsPerBlock = 1; \
                helpers::GpuTimer timer1(stream, "Timer_" + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                for(int i = 0; i < timingLoopIters; i++){ \
                    hipMemsetAsync(d_tempH.data(), 0, d_tempH.size() * sizeof(short2), stream); CUERR; \
                    hipMemsetAsync(d_tempE.data(), 0, d_tempE.size() * sizeof(short2), stream); CUERR; \
                    old::NW_local_affine_read4_float_query_Protein<groupsize, numRegs><<<SDIV(numSubjects, alignmentsPerBlock), blocksize, 0, stream>>>( \
                        d_subjects.data(),  \
                        d_scores_vec[i].data(),  \
                        (short2*)d_tempH.data(), \
                        (short2*)d_tempE.data(), \
                        d_subjectOffsets.data(),  \
                        d_subjectLengths_sizet.data(),  \
                        d_selectedPositions_sizet.data(),  \
                        queryLength,  \
                        gop,  \
                        gex \
                    ); CUERR \
                } \
                timer1.stop(); \
                double gcups = timingCups / 1000. / 1000. / 1000.; \
                gcups = gcups / (timer1.elapsed() / 1000); \
                gcupsVec.push_back(std::make_tuple(gcups,blocksize,groupsize, numRegs )); \
            }


            #define runManyPassFloat_new(blocksize, groupsize, numRegs){ \
                assert(groupsize == 32); \
                assert(blocksize == 32); \
                const char4* d_query4 = (const char4*)d_query.data(); \
                helpers::GpuTimer timer1(stream, "Timer_" + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                for(int i = 0; i < timingLoopIters; i++){ \
                    hipMemsetAsync(d_tempH.data(), 0, d_tempH.size() * sizeof(short2), stream); CUERR; \
                    hipMemsetAsync(d_tempE.data(), 0, d_tempE.size() * sizeof(short2), stream); CUERR; \
                    cudasw4::call_NW_local_affine_read4_float_query_Protein_new<numRegs>( \
                        blosumType, \
                        d_subjects.data(),  \
                        d_scores_vec[i].data(),  \
                        d_tempH.data(), \
                        d_tempE.data(), \
                        d_subjectOffsets.data(),  \
                        d_subjectLengths_lengtht.data(),  \
                        d_selectedPositions_reft.data(),  \
                        numSubjects, \
                        d_query4, \
                        queryLength,  \
                        gop,  \
                        gex, \
                        stream \
                    ); CUERR \
                } \
                timer1.stop(); \
                double gcups = timingCups / 1000. / 1000. / 1000.; \
                gcups = gcups / (timer1.elapsed() / 1000); \
                gcupsVec.push_back(std::make_tuple(gcups,blocksize,groupsize, numRegs )); \
            }

            #define compareManyPassFloatNew(blocksize, groupsize, numRegs){ \
                assert(groupsize == 32); \
                assert(blocksize == 32); \
                const char4* d_query4 = (const char4*)d_query.data(); \
                constexpr int alignmentsPerBlock = 1; \
                    hipMemsetAsync(d_tempH.data(), 0, d_tempH.size() * sizeof(short2), stream); CUERR; \
                    hipMemsetAsync(d_tempE.data(), 0, d_tempE.size() * sizeof(short2), stream); CUERR; \
                    helpers::GpuTimer timer1(stream, "old " + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                    old::NW_local_affine_read4_float_query_Protein<groupsize, numRegs><<<SDIV(numSubjects, alignmentsPerBlock), blocksize, 0, stream>>>( \
                        d_subjects.data(),  \
                        d_scores_vec[0].data(),  \
                        (short2*)d_tempH.data(), \
                        (short2*)d_tempE.data(), \
                        d_subjectOffsets.data(),  \
                        d_subjectLengths_sizet.data(),  \
                        d_selectedPositions_sizet.data(),  \
                        queryLength,  \
                        gop,  \
                        gex \
                    ); CUERR \
                    timer1.printGCUPS(((double(queryLength) * pseudodbSeqLength * numSubjects)));\
                    hipMemsetAsync(d_tempH.data(), 0, d_tempH.size() * sizeof(short2), stream); CUERR; \
                    hipMemsetAsync(d_tempE.data(), 0, d_tempE.size() * sizeof(short2), stream); CUERR; \
                    helpers::GpuTimer timer2(stream, "new " + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                    cudasw4::call_NW_local_affine_read4_float_query_Protein_new<numRegs>( \
                        blosumType, \
                        d_subjects.data(),  \
                        d_scores_vec[1].data(),  \
                        d_tempH.data(), \
                        d_tempE.data(), \
                        d_subjectOffsets.data(),  \
                        d_subjectLengths_lengtht.data(),  \
                        d_selectedPositions_reft.data(),  \
                        numSubjects, \
                        d_query4, \
                        queryLength,  \
                        gop,  \
                        gex, \
                        stream \
                    ); CUERR \
                    timer2.printGCUPS(((double(queryLength) * pseudodbSeqLength * numSubjects)));\
                checkIfEqualResultsNew(); \
            }

            compareManyPassFloatNew(32, 32, 12);

            // std::cout << "start 4\n"; runManyPassFloat(32, 32, 4);
            // std::cout << "start 6\n"; runManyPassFloat(32, 32, 6);
            // std::cout << "start 8\n"; runManyPassFloat(32, 32, 8);
            // std::cout << "start 10\n"; runManyPassFloat(32, 32, 10);
            // std::cout << "start 12\n"; runManyPassFloat(32, 32, 12);
            // std::cout << "start 14\n"; runManyPassFloat(32, 32, 14);
            // std::cout << "start 16\n"; runManyPassFloat(32, 32, 16);
            // runManyPassFloat(32, 32, 2);
            // runManyPassFloat(32, 32, 4);
            // runManyPassFloat(32, 32, 6);
            // runManyPassFloat(32, 32, 8);
            // runManyPassFloat(32, 32, 10);
            // runManyPassFloat(32, 32, 12);
            // runManyPassFloat(32, 32, 14);
            // runManyPassFloat(32, 32, 16);
            // runManyPassFloat(32, 32, 18);
            runManyPassFloat(32, 32, 20);
            // runManyPassFloat(32, 32, 22);
            // runManyPassFloat(32, 32, 24);
            // runManyPassFloat(32, 32, 26);
            // runManyPassFloat(32, 32, 28);
            // runManyPassFloat(32, 32, 30);
            // runManyPassFloat(32, 32, 32);

            std::sort(gcupsVec.begin(), gcupsVec.end(), [](const auto& l, const auto& r){ return std::get<0>(l) > std::get<0>(r);});
            //for(int i = 0; i < std::min(3, int(gcupsVec.size())); i++){
            for(int i = 0; i < int(gcupsVec.size()); i++){
                GCUPSstats data = gcupsVec[i];
                std::cout << std::get<0>(data) << " GCUPS, " << std::get<1>(data) << " " << std::get<2>(data) << " " << std::get<3>(data) << "\n";
            }
            std::cout << "\n";
            gcupsVec.clear();

            // runManyPassFloat_new(32, 32, 6);
            // runManyPassFloat_new(32, 32, 8);
            // runManyPassFloat_new(32, 32, 10);
            // runManyPassFloat_new(32, 32, 12);
            // runManyPassFloat_new(32, 32, 14);
            // runManyPassFloat_new(32, 32, 16);
            // runManyPassFloat_new(32, 32, 18);
            runManyPassFloat_new(32, 32, 20);
            // runManyPassFloat_new(32, 32, 22);
            // runManyPassFloat_new(32, 32, 24);
            // runManyPassFloat_new(32, 32, 26);
            // runManyPassFloat_new(32, 32, 28);
            // runManyPassFloat_new(32, 32, 30);
            // runManyPassFloat_new(32, 32, 32);

            std::sort(gcupsVec.begin(), gcupsVec.end(), [](const auto& l, const auto& r){ return std::get<0>(l) > std::get<0>(r);});
            //for(int i = 0; i < std::min(3, int(gcupsVec.size())); i++){
            for(int i = 0; i < int(gcupsVec.size()); i++){
                GCUPSstats data = gcupsVec[i];
                std::cout << std::get<0>(data) << " GCUPS, " << std::get<1>(data) << " " << std::get<2>(data) << " " << std::get<3>(data) << "\n";
            }
            std::cout << "\n";
            gcupsVec.clear();
        }

    #endif




    // single pass dpx s16
    #if 0
        std::cout << "NW_local_affine_single_pass_s16_DPX\n";

        //for(int pseudodbSeqLength : {48, 64, 80, 96, 112, 128, 144, 160, 176, 192, 208, 224, 240, 256, 288, 320, 352, 384, 416, 448, 480, 512, 576, 640, 704, 768, 832, 896, 960, 1024, 1088, 1152, 1216, 1280}){
        //for(int pseudodbSeqLength : {64, 128, 192, 256, 320, 384, 448, 512, 576, 640, 704, 768, 832, 896, 960, 1024}){
        //for(int pseudodbSeqLength : {192}){
        //for(int pseudodbSeqLength = 11; pseudodbSeqLength <= 64; pseudodbSeqLength++){
        {
            const int pseudodbSeqLength = pseudolength;
            std::cout << "pseudodbSeqLength: " << pseudodbSeqLength << "\n";
    
            cudasw4::PseudoDB fullDB = cudasw4::loadPseudoDB(numSubjects, pseudodbSeqLength);
            const auto& dbData = fullDB.getData();

            std::vector<MyDeviceBuffer<float>> d_scores_vec(std::max(2, timingLoopIters));
            std::vector<MyDeviceBuffer<cudasw4::ReferenceIdT>> d_overflow_positions_vec_reft(std::max(2, timingLoopIters));
            std::vector<MyDeviceBuffer<size_t>> d_overflow_positions_vec_sizet(std::max(2, timingLoopIters));
            std::vector<MyDeviceBuffer<int>> d_overflow_number_vec(std::max(2, timingLoopIters));
            for(int i = 0; i < std::max(2, timingLoopIters); i++){
                d_scores_vec[i].resize(numSubjects);
                d_overflow_positions_vec_reft[i].resize(numSubjects);
                d_overflow_positions_vec_sizet[i].resize(numSubjects);
                d_overflow_number_vec[i].resize(1);
                hipMemsetAsync(d_overflow_number_vec[i].data(), 0, sizeof(int), stream);
            }
            MyDeviceBuffer<cudasw4::ReferenceIdT> d_selectedPositions_reft(numSubjects);
            MyDeviceBuffer<size_t> d_selectedPositions_sizet(numSubjects);
            thrust::sequence(thrust::cuda::par_nosync.on(stream), d_selectedPositions_reft.begin(), d_selectedPositions_reft.end(), cudasw4::ReferenceIdT(0));
            thrust::sequence(thrust::cuda::par_nosync.on(stream), d_selectedPositions_sizet.begin(), d_selectedPositions_sizet.end(), size_t(0));

            MyDeviceBuffer<char> d_subjects(dbData.numChars());
            MyDeviceBuffer<size_t> d_subjectOffsets(numSubjects+1);
            MyDeviceBuffer<cudasw4::SequenceLengthT> d_subjectLengths_lengtht(numSubjects);
            MyDeviceBuffer<size_t> d_subjectLengths_sizet(numSubjects);

            hipMemcpyAsync(d_subjects.data(), dbData.chars(), dbData.numChars(), H2D, stream); CUERR;
            hipMemcpyAsync(d_subjectOffsets.data(), dbData.offsets(), sizeof(size_t) * (numSubjects+1), H2D, stream); CUERR;
            hipMemcpyAsync(d_subjectLengths_lengtht.data(), dbData.lengths(), sizeof(cudasw4::SequenceLengthT) * numSubjects, H2D, stream); CUERR;

            thrust::copy(
                thrust::cuda::par.on(stream),
                d_subjectLengths_lengtht.data(),
                d_subjectLengths_lengtht.data() + numSubjects,
                d_subjectLengths_sizet.data()
            );

            auto checkIfEqualResultsNew = [&](){
                const float overflowscore = 123456;
                auto overflowiter = thrust::make_constant_iterator(overflowscore);
                for(int i = 0; i < 2; i++){
                    int numOverflow = 0;
                    hipMemcpyAsync(&numOverflow, d_overflow_number_vec[i].data(), sizeof(int), D2H, stream); CUERR;
                    hipStreamSynchronize(stream); CUERR;
                    // if(i == 0){
                    //     std::cout << "Num overflows: " << numOverflow << "\n";
                    // }
                    thrust::scatter(
                        thrust::cuda::par_nosync.on(stream),
                        overflowiter,
                        overflowiter + numOverflow,
                        d_overflow_positions_vec_reft[i].data(),
                        d_scores_vec[i].data()
                    );
                }
                for(int i = 1; i < 2; i++){
                    bool equal = thrust::equal(
                        thrust::cuda::par_nosync.on(stream),
                        d_scores_vec[i].data(),
                        d_scores_vec[i].data() + numSubjects,
                        d_scores_vec[0].data()
                    );
                    if(!equal){
                        std::cout << "i = " << i << ", scores not equal\n";
                    }else{
                        std::cout << "ok\n";
                    }
                }
            };

            const double timingCups = ((double(queryLength) * pseudodbSeqLength * numSubjects)) * timingLoopIters;

            using GCUPSstats = std::tuple<double, int, int, int>;

            std::vector<GCUPSstats> gcupsVec;

            #define runSinglePassDPX_s16(blocksize, groupsize, numRegs){ \
                assert(blocksize % groupsize == 0); \
                if(pseudodbSeqLength <= groupsize * numRegs){ \
                    constexpr int alignmentsPerBlock = (blocksize / groupsize) * 2; \
                    helpers::GpuTimer timer1(stream, "Timer_" + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                    for(int i = 0; i < timingLoopIters; i++){ \
                        old::NW_local_affine_single_pass_s16_DPX<groupsize, numRegs><<<SDIV(numSubjects, alignmentsPerBlock), blocksize, 0, stream>>>( \
                            d_subjects.data(),  \
                            d_scores_vec[i].data(),  \
                            d_subjectOffsets.data(),  \
                            d_subjectLengths_sizet.data(),  \
                            d_selectedPositions_sizet.data(),  \
                            numSubjects,  \
                            queryLength,  \
                            gop,  \
                            gex \
                        ); CUERR \
                    } \
                    double gcups = timingCups / 1000. / 1000. / 1000.; \
                    gcups = gcups / (timer1.elapsed() / 1000); \
                    gcupsVec.push_back(std::make_tuple(gcups,blocksize,groupsize, numRegs )); \
                } \
            }
            #define runSinglePassDPX_s16_new(blocksize, groupsize, numRegs){ \
                assert(blocksize % groupsize == 0); \
                const char4* d_query4 = (const char4*)d_query.data(); \
                if(pseudodbSeqLength <= groupsize * numRegs){ \
                    helpers::GpuTimer timer1(stream, "Timer_" + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                    for(int i = 0; i < timingLoopIters; i++){ \
                        cudasw4::call_NW_local_affine_single_pass_s16_DPX_new<blocksize, groupsize, numRegs>( \
                            blosumType, \
                            d_subjects.data(),  \
                            d_scores_vec[i].data(),  \
                            d_subjectOffsets.data(),  \
                            d_subjectLengths_lengtht.data(),  \
                            d_selectedPositions_reft.data(),  \
                            numSubjects,  \
                            d_overflow_positions_vec_reft[i].data(),  \
                            d_overflow_number_vec[i].data(),  \
                            0,  \
                            d_query4, \
                            queryLength,  \
                            gop,  \
                            gex, \
                            stream \
                        ); CUERR \
                    } \
                    double gcups = timingCups / 1000. / 1000. / 1000.; \
                    gcups = gcups / (timer1.elapsed() / 1000); \
                    gcupsVec.push_back(std::make_tuple(gcups,blocksize,groupsize, numRegs )); \
                } \
            }

            #define compareSinglePassDPX_s16New(blocksize, groupsize, numRegs){ \
                assert(blocksize % groupsize == 0); \
                const char4* d_query4 = (const char4*)d_query.data(); \
                constexpr int alignmentsPerBlock = (blocksize / groupsize) * 2; \
                    helpers::GpuTimer timer1(stream, "old " + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                    old::NW_local_affine_single_pass_s16_DPX<groupsize, numRegs><<<SDIV(numSubjects, alignmentsPerBlock), blocksize, 0, stream>>>( \
                        d_subjects.data(),  \
                        d_scores_vec[0].data(),  \
                        d_subjectOffsets.data(),  \
                        d_subjectLengths_sizet.data(),  \
                        d_selectedPositions_sizet.data(),  \
                        numSubjects,  \
                        queryLength,  \
                        gop,  \
                        gex \
                    ); CUERR \
                    timer1.printGCUPS(((double(queryLength) * pseudodbSeqLength * numSubjects)));\
                    helpers::GpuTimer timer2(stream, "new " + std::to_string(blocksize) + "_" + std::to_string(groupsize) + "_" + std::to_string(numRegs)); \
                    cudasw4::call_NW_local_affine_single_pass_s16_DPX_new<blocksize, groupsize, numRegs>( \
                        blosumType, \
                        d_subjects.data(),  \
                        d_scores_vec[1].data(),  \
                        d_subjectOffsets.data(),  \
                        d_subjectLengths_lengtht.data(),  \
                        d_selectedPositions_reft.data(),  \
                        numSubjects,  \
                        d_overflow_positions_vec_reft[1].data(),  \
                        d_overflow_number_vec[1].data(),  \
                        0,  \
                        d_query4, \
                        queryLength,  \
                        gop,  \
                        gex, \
                        stream \
                    ); CUERR \
                    timer2.printGCUPS(((double(queryLength) * pseudodbSeqLength * numSubjects)));\
                checkIfEqualResultsNew(); \
            }

            #define runSinglePassDPX_s16_numregs(blocksize, numRegs){ \
                runSinglePassDPX_s16(blocksize, 1, numRegs); \
                runSinglePassDPX_s16(blocksize, 2, numRegs); \
                runSinglePassDPX_s16(blocksize, 4, numRegs); \
                runSinglePassDPX_s16(blocksize, 8, numRegs); \
                runSinglePassDPX_s16(blocksize, 16, numRegs); \
                runSinglePassDPX_s16(blocksize, 32, numRegs); \
            }
            #define runSinglePassDPX_s16_numregs_new(blocksize, numRegs){ \
                runSinglePassDPX_s16_new(blocksize, 1, numRegs); \
                runSinglePassDPX_s16_new(blocksize, 2, numRegs); \
                runSinglePassDPX_s16_new(blocksize, 4, numRegs); \
                runSinglePassDPX_s16_new(blocksize, 8, numRegs); \
                runSinglePassDPX_s16_new(blocksize, 16, numRegs); \
                runSinglePassDPX_s16_new(blocksize, 32, numRegs); \
            }

            compareSinglePassDPX_s16New(256, 32, 16);

            // runSinglePassDPX_s16_numregs(256, 32);
            // runSinglePassDPX_s16_numregs(256, 30);
            // runSinglePassDPX_s16_numregs(256, 28);
            // runSinglePassDPX_s16_numregs(256, 26);
            // runSinglePassDPX_s16_numregs(256, 24);
            // runSinglePassDPX_s16_numregs(256, 22);
            // runSinglePassDPX_s16_numregs(256, 20);
            // runSinglePassDPX_s16_numregs(256, 18);
            // runSinglePassDPX_s16_numregs(256, 16);
            // runSinglePassDPX_s16_numregs(256, 14);
            // runSinglePassDPX_s16_numregs(256, 12);
            // runSinglePassDPX_s16_numregs(256, 10);
            // runSinglePassDPX_s16_numregs(256, 8);
            // runSinglePassDPX_s16_numregs(256, 6);
            // runSinglePassDPX_s16_numregs(256, 4);
            // runSinglePassDPX_s16_numregs(256, 2);

            // runSinglePassDPX_s16(256,2,24);
            // runSinglePassDPX_s16(256,4,16);
            // runSinglePassDPX_s16(256,8,10);
            // runSinglePassDPX_s16(256,8,12);
            // runSinglePassDPX_s16(256,8,14);
            // runSinglePassDPX_s16(256,8,16);
            // runSinglePassDPX_s16(256,8,18);
            // runSinglePassDPX_s16(256,8,20);
            // runSinglePassDPX_s16(256,8,22);
            // runSinglePassDPX_s16(256,8,24);
            // runSinglePassDPX_s16(256,8,26);
            // runSinglePassDPX_s16(256,8,28);
            // runSinglePassDPX_s16(256,8,30);
            // runSinglePassDPX_s16(256,8,32);
            // runSinglePassDPX_s16(256,16,18);
            // runSinglePassDPX_s16(256,16,20);
            // runSinglePassDPX_s16(256,16,22);
            // runSinglePassDPX_s16(256,16,24);
            // runSinglePassDPX_s16(256,16,26);
            // runSinglePassDPX_s16(256,16,28);
            // runSinglePassDPX_s16(256,16,30);
            // runSinglePassDPX_s16(256,16,32);
            // runSinglePassDPX_s16(256,32,18);
            // runSinglePassDPX_s16(256,32,20);
            // runSinglePassDPX_s16(256,32,22);
            // runSinglePassDPX_s16(256,32,24);
            // runSinglePassDPX_s16(256,32,26);
            // runSinglePassDPX_s16(256,32,28);
            // runSinglePassDPX_s16(256,32,30);
            // runSinglePassDPX_s16(256,32,32);
            // runSinglePassDPX_s16(256,32,34);
            // runSinglePassDPX_s16(256,32,36);
            // runSinglePassDPX_s16(256,32,38);
            // runSinglePassDPX_s16(256,32,40);


            runSinglePassDPX_s16(256,8,16);
            runSinglePassDPX_s16(256,8,32);
            runSinglePassDPX_s16(256,16,32);
            runSinglePassDPX_s16(256,32,24);
            runSinglePassDPX_s16(256,32,32);

            std::sort(gcupsVec.begin(), gcupsVec.end(), [](const auto& l, const auto& r){ return std::get<0>(l) > std::get<0>(r);});

            std::cout << "old\n";
            //for(int i = 0; i < std::min(3, int(gcupsVec.size())); i++){
            for(int i = 0; i < int(gcupsVec.size()); i++){
                GCUPSstats data = gcupsVec[i];
                std::cout << std::get<0>(data) << " GCUPS, " << std::get<1>(data) << " " << std::get<2>(data) << " " << std::get<3>(data) << "\n";
            }
            gcupsVec.clear();
            

            // runSinglePassDPX_s16_numregs_new(256, 32);
            // runSinglePassDPX_s16_numregs_new(256, 30);
            // runSinglePassDPX_s16_numregs_new(256, 28);
            // runSinglePassDPX_s16_numregs_new(256, 26);
            // runSinglePassDPX_s16_numregs_new(256, 24);
            // runSinglePassDPX_s16_numregs_new(256, 22);
            // runSinglePassDPX_s16_numregs_new(256, 20);
            // runSinglePassDPX_s16_numregs_new(256, 18);
            // runSinglePassDPX_s16_numregs_new(256, 16);
            // runSinglePassDPX_s16_numregs_new(256, 14);
            // runSinglePassDPX_s16_numregs_new(256, 12);
            // runSinglePassDPX_s16_numregs_new(256, 10);
            // runSinglePassDPX_s16_numregs_new(256, 8);
            // runSinglePassDPX_s16_numregs_new(256, 6);
            // runSinglePassDPX_s16_numregs_new(256, 4);
            // runSinglePassDPX_s16_numregs_new(256, 2);

            // runSinglePassDPX_s16_new(256,2,24);
            // runSinglePassDPX_s16_new(256,4,16);
            // runSinglePassDPX_s16_new(256,8,10);
            // runSinglePassDPX_s16_new(256,8,12);
            // runSinglePassDPX_s16_new(256,8,14);
            // runSinglePassDPX_s16_new(256,8,16);
            // runSinglePassDPX_s16_new(256,8,18);
            // runSinglePassDPX_s16_new(256,8,20);
            // runSinglePassDPX_s16_new(256,8,22);
            // runSinglePassDPX_s16_new(256,8,24);
            // runSinglePassDPX_s16_new(256,8,26);
            // runSinglePassDPX_s16_new(256,8,28);
            // runSinglePassDPX_s16_new(256,8,30);
            // runSinglePassDPX_s16_new(256,8,32);
            // runSinglePassDPX_s16_new(256,16,18);
            // runSinglePassDPX_s16_new(256,16,20);
            // runSinglePassDPX_s16_new(256,16,22);
            // runSinglePassDPX_s16_new(256,16,24);
            // runSinglePassDPX_s16_new(256,16,26);
            // runSinglePassDPX_s16_new(256,16,28);
            // runSinglePassDPX_s16_new(256,16,30);
            // runSinglePassDPX_s16_new(256,16,32);
            // runSinglePassDPX_s16_new(256,32,18);
            // runSinglePassDPX_s16_new(256,32,20);
            // runSinglePassDPX_s16_new(256,32,22);
            // runSinglePassDPX_s16_new(256,32,24);
            // runSinglePassDPX_s16_new(256,32,26);
            // runSinglePassDPX_s16_new(256,32,28);
            // runSinglePassDPX_s16_new(256,32,30);
            // runSinglePassDPX_s16_new(256,32,32);
            // runSinglePassDPX_s16_new(256,32,34);
            // runSinglePassDPX_s16_new(256,32,36);
            // runSinglePassDPX_s16_new(256,32,38);
            // runSinglePassDPX_s16_new(256,32,40);

            runSinglePassDPX_s16_new(256,8,16);
            runSinglePassDPX_s16_new(256,8,32);
            runSinglePassDPX_s16_new(256,16,32);
            runSinglePassDPX_s16_new(256,32,24);
            runSinglePassDPX_s16_new(256,32,32);

            std::sort(gcupsVec.begin(), gcupsVec.end(), [](const auto& l, const auto& r){ return std::get<0>(l) > std::get<0>(r);});

            std::cout << "new\n";
            //for(int i = 0; i < std::min(3, int(gcupsVec.size())); i++){
            for(int i = 0; i < int(gcupsVec.size()); i++){
                GCUPSstats data = gcupsVec[i];
                std::cout << std::get<0>(data) << " GCUPS, " << std::get<1>(data) << " " << std::get<2>(data) << " " << std::get<3>(data) << "\n";
            }
            gcupsVec.clear();
        }

    #endif
}